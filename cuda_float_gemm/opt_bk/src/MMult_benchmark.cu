#include "params.h"
#include "utils.h"
#include "MMult.h"

// void MMult_benchmark(hipblasHandle_t handle, int m, int k, int n, float *d_A, float *d_B, float *d_C, int lda, int ldb, int ldc) {
//     float alpha = 1.0, beta = 0.0;
//     checkCuBlasErrors (
//         // https://blog.csdn.net/u011197534/article/details/78378536
//         hipblasSgemm (handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
//             m, n, k, &alpha, 
//             d_A, k, d_B, n, &beta, d_C, m
//         )
//     );
// }

// Note that hipblasSgemm works in col-major, so we calculate by CT = BT * AT
// https://blog.csdn.net/u011197534/article/details/78378536
// https://blog.csdn.net/HaoBBNuanMM/article/details/103054357
void MMult_benchmark(hipblasHandle_t handle, int m, int k, int n, float *d_A, float *d_B, float *d_C, int lda, int ldb, int ldc) {
    float alpha = 1.0, beta = 0.0;
    checkCuBlasErrors (
        hipblasSgemm(handle, 
                    HIPBLAS_OP_N, 
                    HIPBLAS_OP_N, 
                    n,
                    m, 
                    k,
                    &alpha, 
                    d_B, 
                    n,
                    d_A,
                    k, 
                    &beta, 
                    d_C, 
                    n
        )
    );
}